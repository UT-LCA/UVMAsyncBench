#include "hip/hip_runtime.h"
/**
 * gemm.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "../../../common/cupti_add.h"
#include "../../../common/cpu_timestamps.h"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>

using namespace nvcuda::experimental;

#define PREFETCH_COUNT 2

#define SMALL_FLOAT_VAL 0.00000001f

double rtclock()
{
	struct timezone Tzp;
	struct timeval Tp;
	uint64_t stat;
	stat = gettimeofday(&Tp, &Tzp);
	if (stat != 0)
		printf("Error return from gettimeofday: %d", stat);
	return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

float absVal(float a)
{
	if (a < 0)
	{
		return (a * -1);
	}
	else
	{
		return a;
	}
}

float percentDiff(double val1, double val2)
{
	if ((absVal(val1) < 0.01) && (absVal(val2) < 0.01))
	{
		return 0.0f;
	}

	else
	{
		return 100.0f * (absVal(absVal(val1 - val2) / absVal(val1 + SMALL_FLOAT_VAL)));
	}
}

#define GPU_DEVICE 5

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.005

/* Problem size */
#define SIZE 1073741824
#define ITER 100
uint64_t NI;

/* Thread block dimensions */
#ifndef DIM_THREAD_BLOCK
#define DIM_THREAD_BLOCK 256
#endif

#ifndef BATCH_SIZE
#define BATCH_SIZE 16
#endif

#ifndef NBLOCKS
#define NBLOCKS 64
#endif

#define LCG_A 1.1f
#define LCG_B 1.1f

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;
// typedef uint64_t DATA_TYPE;

void saxpy(DATA_TYPE *A, uint64_t iterations)
{	
	for (uint64_t i = 0; i < NI; i++) {
		for (uint64_t iter = 0; iter < iterations; iter++) {
			A[i] = LCG_A * A[i] + LCG_B;
		}
	}
}

void initCPU(DATA_TYPE *A)
{
  	for (uint64_t i = 0; i < NI; i++) {
		A[i] = ((DATA_TYPE) i) / NI;
	}
}

void initGPU(DATA_TYPE *A_gpu)
{
  	for (uint64_t i = 0; i < NI; i++) {
		A_gpu[i] = ((DATA_TYPE)i) / NI;
	}
}


void compareResults(DATA_TYPE* A, DATA_TYPE* A_outputFromGpu)
{
	uint64_t fail = 0;
	
	// Compare C1 and C2
	for (uint64_t i = 0; i < NI; i++) {
		// printf("%lld, GPU is %f, CPU is %f.\n", i, A[i], A_outputFromGpu[i]);
		if (percentDiff(A[i], A_outputFromGpu[i]) > PERCENT_DIFF_ERROR_THRESHOLD) {
			fail++;
			printf("%lld, GPU is %f, CPU is %f.\n", i, A[i], A_outputFromGpu[i]);
		}
	}
	
	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}

__global__ void vector_seq_kernel(DATA_TYPE *a, uint64_t NI, uint64_t iterations, uint64_t block_size)
{
	cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
	pipeline pipe;

	const uint64_t mem_size = DIM_THREAD_BLOCK * BATCH_SIZE;

	// __shared__ DATA_TYPE tmp[mem_size * PREFETCH_COUNT];
	extern __shared__ DATA_TYPE tmp[];

	uint64_t total_tiles = NI / mem_size;
	uint64_t base_tiles = total_tiles / gridDim.x;

	uint64_t tiles_this_block = block_size / mem_size;

	uint64_t fetch = base_tiles * blockIdx.x;
	uint64_t end_tile = fetch + tiles_this_block;

	for (uint64_t compute = fetch; compute < end_tile; compute++)
	{
		for (; fetch < end_tile && fetch < compute + PREFETCH_COUNT; fetch++)
		{
			for (uint64_t i = threadIdx.x; i < mem_size; i += blockDim.x)
			{
				memcpy_async(tmp[(fetch % PREFETCH_COUNT) * mem_size + i], a[fetch * mem_size + i], pipe);
			}
			pipe.commit();
		}
		if (fetch == end_tile)
		{
			for (uint64_t i = 0; i < PREFETCH_COUNT - 1; ++i)
			{
				pipe.commit();
			}
			++fetch;
		}
		pipe.wait_prior<PREFETCH_COUNT - 1>();
		block.sync();

		for (uint64_t i = threadIdx.x; i < mem_size; i += blockDim.x)
		{
			for (uint64_t iter = 0; iter < iterations; iter++)
			{
				tmp[(compute % PREFETCH_COUNT) * mem_size + i] = LCG_A * tmp[(compute % PREFETCH_COUNT) * mem_size + i] + LCG_B;
			}
		}
		block.sync();

		for (uint64_t i = threadIdx.x; i < mem_size; i += blockDim.x)
		{
			a[compute * mem_size + i] = tmp[(compute % PREFETCH_COUNT) * mem_size + i];
		}
		block.sync();
	}
}

void saxpyCuda(DATA_TYPE *A, DATA_TYPE *A_gpu, uint64_t iterations, uint64_t block_size)
{
	double t_start, t_end;
	if (block_size <= DIM_THREAD_BLOCK)
		block_size = DIM_THREAD_BLOCK;

	dim3 block(DIM_THREAD_BLOCK);
	dim3 grid(NI / block_size);

	int MaxBytesofSharedMemory = DIM_THREAD_BLOCK * BATCH_SIZE * PREFETCH_COUNT * sizeof(DATA_TYPE);
	hipFuncSetAttribute(reinterpret_cast<const void*>(vector_seq_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, MaxBytesofSharedMemory);

	//t_start = rtclock();
	hipStream_t stream1;
	hipStreamCreate(&stream1);

	hipMemPrefetchAsync(A_gpu, NI * sizeof(DATA_TYPE), GPU_DEVICE, stream1);
	hipStreamSynchronize(stream1);
	vector_seq_kernel<<<grid, block, MaxBytesofSharedMemory, stream1>>>(A_gpu, NI, iterations, block_size);
	hipDeviceSynchronize();

	// vector_seq_kernel<<<grid, block>>>(A_gpu, NI, iterations, block_size);
	// hipDeviceSynchronize();
	//t_end = rtclock();

	//fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);
}

extern inline __attribute__((always_inline)) unsigned long rdtsc()
{
           unsigned long a, d;

              __asm__ volatile("rdtsc" : "=a" (a), "=d" (d));

                 return (a | (d << 32));
}

extern inline __attribute__((always_inline)) unsigned long rdtsp() {
                struct timespec tms;
                    if (clock_gettime(CLOCK_REALTIME, &tms)) {
                                    return -1;
                                        }
                        unsigned long ns = tms.tv_sec * 1000000000;
                            ns += tms.tv_nsec;
                                return ns;
}

int main(int argc, char *argv[])
{
	uint64_t start_tsc = rdtsc();
	uint64_t start_tsp = rdtsp();
	printf("start_tsc %lu start_tsp %lu\n", start_tsc, start_tsp);
	uint64_t iterations = ITER;
	uint64_t block_size = DIM_THREAD_BLOCK * BATCH_SIZE;
	if (argc >= 4) {
		NI = atoll(argv[1]);
		iterations = atoi(argv[2]);
		block_size = atoi(argv[3]);
	}
	else {
		NI = SIZE;
		iterations = ITER;
		block_size = DIM_THREAD_BLOCK * BATCH_SIZE;
	}

	int nblocks = NBLOCKS;
	block_size = NI / nblocks;

	double t_start, t_end;

	DATA_TYPE* A;
	DATA_TYPE *A_gpu;

	A = (DATA_TYPE*)malloc(NI*sizeof(DATA_TYPE));

	initCPU(A);
	GPU_argv_init();

	initTrace();
	startCPU();

	hipMallocManaged(&A_gpu, sizeof(DATA_TYPE) * NI);

	// initGPU(A_gpu);
	// overlapStartCPU();
	memcpy(A_gpu, A, NI * sizeof(DATA_TYPE));
	// overlapEndCPU();

	saxpyCuda(A, A_gpu, iterations, block_size);
	memcpy(A, A_gpu, NI * sizeof(DATA_TYPE));

	// t_start = rtclock();	
	// saxpy(A, iterations);
	// t_end = rtclock();
	// fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);

	// compareResults(A_gpu, A);
	hipFree(A_gpu);
	endCPU();
	finiTrace();

	
	free(A);
    return 0;
}

