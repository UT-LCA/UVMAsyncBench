#include "hip/hip_runtime.h"
/**
 * gemm.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "../../../common/cupti_add.h"
#include "../../../common/cpu_timestamps.h"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>

using namespace nvcuda::experimental;

#define PREFETCH_COUNT 2

#define SMALL_FLOAT_VAL 0.00000001f

double rtclock()
{
	struct timezone Tzp;
	struct timeval Tp;
	uint64_t stat;
	stat = gettimeofday(&Tp, &Tzp);
	if (stat != 0)
		printf("Error return from gettimeofday: %d", stat);
	return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

float absVal(float a)
{
	if (a < 0)
	{
		return (a * -1);
	}
	else
	{
		return a;
	}
}

float percentDiff(double val1, double val2)
{
	if ((absVal(val1) < 0.01) && (absVal(val2) < 0.01))
	{
		return 0.0f;
	}

	else
	{
		return 100.0f * (absVal(absVal(val1 - val2) / absVal(val1 + SMALL_FLOAT_VAL)));
	}
}

#define GPU_DEVICE 5

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

/* Problem size */
#define SIZE 4096000
#define ITER 100
uint64_t NI;

/* Thread block dimensions */
#define DIM_THREAD_BLOCK 256

#define BATCH_SIZE 8

/* Declared constant values for ALPHA and BETA (same as values in PolyBench 2.0) */
#define ALPHA 1.1f
#define BETA 1.1f

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;
// typedef uint64_t DATA_TYPE;

void saxpy(DATA_TYPE *A, DATA_TYPE *B, uint64_t iterations)
{	
	for (uint64_t i = 0; i < NI; i++) {
		for (uint64_t iter = 0; iter < iterations; iter++) {
			B[i] = ALPHA * A[i] + B[i];
		}
	}
}

void initGPU(DATA_TYPE *A_gpu, DATA_TYPE *B_gpu)
{
	for (uint64_t i = 0; i < NI; i++) {
		A_gpu[i] = ((DATA_TYPE)i) / NI;
	}
  for (uint64_t i = 0; i < NI; i++) {
		B_gpu[i] = ((DATA_TYPE)i + 2) / NI;
	}
    	
}

void initCPU(DATA_TYPE *A, DATA_TYPE *B)
{
	for (uint64_t i = 0; i < NI; i++) {
		A[i] = ((DATA_TYPE)i) / NI;
	}
  for (uint64_t i = 0; i < NI; i++) {
		B[i] = ((DATA_TYPE)i + 2) / NI;
	}   	
}

void compareResults(DATA_TYPE* B, DATA_TYPE* B_outputFromGpu)
{
	uint64_t fail = 0;
	
	// Compare C1 and C2
	for (uint64_t i = 0; i < NI; i++) {
		// printf("%lld, GPU is %f, CPU is %f.\n", i, B[i], B_outputFromGpu[i]);
		if (percentDiff(B[i], B_outputFromGpu[i]) > PERCENT_DIFF_ERROR_THRESHOLD) {
			fail++;
			printf("%lld, GPU is %f, CPU is %f.\n", i, B[i], B_outputFromGpu[i]);
		}
	}
	
	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}

__global__ void saxpy_kernel(DATA_TYPE *a, DATA_TYPE *b, uint64_t NI, uint64_t iterations, uint64_t block_size)
{
	cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
	pipeline pipe;

	// Compute each thread's global row and column index
	const uint64_t mem_size = DIM_THREAD_BLOCK * BATCH_SIZE;

	__shared__ DATA_TYPE tmp_a[mem_size * PREFETCH_COUNT];
	__shared__ DATA_TYPE tmp_b[mem_size * PREFETCH_COUNT];

	uint64_t total_tiles = NI / mem_size;
	uint64_t base_tiles = total_tiles / gridDim.x;

	uint64_t tiles_this_block = block_size / mem_size;

	uint64_t fetch = base_tiles * blockIdx.x;
	uint64_t end_tile = fetch + tiles_this_block;

	for (uint64_t compute = fetch; compute < end_tile; compute++)
	{
		for (; fetch < end_tile && fetch < compute + PREFETCH_COUNT; fetch++)
		{
			for (uint64_t i = threadIdx.x; i < mem_size; i += blockDim.x)
			{
				memcpy_async(tmp_a[(fetch % PREFETCH_COUNT) * mem_size + i], a[fetch * mem_size + i], pipe);
				memcpy_async(tmp_b[(fetch % PREFETCH_COUNT) * mem_size + i], b[fetch * mem_size + i], pipe);
			}
			pipe.commit();
		}
		if (fetch == end_tile)
		{
			for (uint64_t i = 0; i < PREFETCH_COUNT - 1; ++i)
			{
				pipe.commit();
			}
			++fetch;
		}
		pipe.wait_prior<PREFETCH_COUNT - 1>();
		block.sync();

		for (uint64_t i = threadIdx.x; i < mem_size; i += blockDim.x)
		{
			for (uint64_t iter = 0; iter < iterations; iter++)
			{
				tmp_b[(compute % PREFETCH_COUNT) * mem_size + i] += ALPHA * tmp_a[(compute % PREFETCH_COUNT) * mem_size + i];
			}
		}
		block.sync();

		for (uint64_t i = threadIdx.x; i < mem_size; i += blockDim.x)
		{
			b[compute * mem_size + i] = tmp_b[(compute % PREFETCH_COUNT) * mem_size + i];
		}
		block.sync();
	}
}

void saxpyCuda(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *A_gpu, DATA_TYPE *B_gpu, uint64_t iterations, uint64_t block_size)
{
	double t_start, t_end;
	if (block_size <= DIM_THREAD_BLOCK)
		block_size = DIM_THREAD_BLOCK;

	dim3 block(DIM_THREAD_BLOCK);
	dim3 grid(NI / block_size);

	//t_start = rtclock();
	hipStream_t stream1;
	hipStream_t stream2;
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);

	hipMemPrefetchAsync(A_gpu, NI * sizeof(DATA_TYPE), GPU_DEVICE, stream1);
	hipStreamSynchronize(stream1);
	hipMemPrefetchAsync(B_gpu, NI * sizeof(DATA_TYPE), GPU_DEVICE, stream2);
	hipStreamSynchronize(stream2);
	saxpy_kernel<<<grid, block, 0, stream2>>>(A_gpu, B_gpu, NI, iterations, block_size);
	hipDeviceSynchronize();

	// saxpy_kernel<<<grid, block>>>(A_gpu, B_gpu, NI, iterations, block_size);
	// hipDeviceSynchronize();
	//t_end = rtclock();

	//fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);   
}

extern inline __attribute__((always_inline)) unsigned long rdtsc()
{
           unsigned long a, d;

              __asm__ volatile("rdtsc" : "=a" (a), "=d" (d));

                 return (a | (d << 32));
}

extern inline __attribute__((always_inline)) unsigned long rdtsp() {
                struct timespec tms;
                    if (clock_gettime(CLOCK_REALTIME, &tms)) {
                                    return -1;
                                        }
                        unsigned long ns = tms.tv_sec * 1000000000;
                            ns += tms.tv_nsec;
                                return ns;
}


int main(int argc, char *argv[])
{
	uint64_t start_tsc = rdtsc();
	uint64_t start_tsp = rdtsp();
	printf("start_tsc %lu start_tsp %lu\n", start_tsc, start_tsp);

	uint64_t iterations = ITER;
	uint64_t block_size = DIM_THREAD_BLOCK * BATCH_SIZE;
	if (argc >= 4) {
		NI = atoll(argv[1]);
		iterations = atoi(argv[2]);
		block_size = atoi(argv[3]);
	} else {
		NI = SIZE;
		iterations = ITER;
		block_size = DIM_THREAD_BLOCK * BATCH_SIZE;
	}

	int nblocks = NI / block_size;
	if (nblocks > 64)
	{
		nblocks = 64;
		block_size = NI / nblocks;
	}

	double t_start, t_end;

	DATA_TYPE* A;
	DATA_TYPE* B;

	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;

	A = (DATA_TYPE*)malloc(NI*sizeof(DATA_TYPE)); 
	B = (DATA_TYPE*)malloc(NI*sizeof(DATA_TYPE));
	
	initCPU(A,B);
	GPU_argv_init();

	initTrace();
	startCPU();

	hipMallocManaged(&A_gpu, sizeof(DATA_TYPE) * NI);
	hipMallocManaged(&B_gpu, sizeof(DATA_TYPE) * NI);

	// initGPU(A_gpu, B_gpu);
	// overlapStartCPU();
	memcpy(A_gpu, A, NI * sizeof(DATA_TYPE));
	memcpy(B_gpu, B, NI * sizeof(DATA_TYPE));
	// overlapEndCPU();

	saxpyCuda(A, B, A_gpu, B_gpu, iterations, block_size);
	memcpy(B, B_gpu, NI * sizeof(DATA_TYPE));

	// t_start = rtclock();	
	// saxpy(A, B, iterations);
	// t_end = rtclock();
	// fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);
	
	// compareResults(B_gpu, B);
	hipFree(A_gpu);
	hipFree(B_gpu);
	endCPU();
	finiTrace();

	free(A);
	free(B);
    return 0;
}
