#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "dropout_layer.h"
#include "cuda_dark.h"
#include "utils.h"
}

__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id < size) input[id] = (rand[id] < prob) ? 0 : input[id]*scale;
}

void forward_dropout_layer_gpu(dropout_layer layer, network net)
{
    if (!net.train) return;
    int size = layer.inputs*layer.batch;
    cuda_random(layer.rand_gpu, size);
    /*
    int i;
    for(i = 0; i < size; ++i){
        layer.rand[i] = rand_uniform();
    }
    cuda_push_array(layer.rand_gpu, layer.rand, size);
    */
    hipStream_t stream1;
    hipStream_t stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    hipMemPrefetchAsync(net.input_gpu, size * sizeof(float), GPU_DEVICE, stream1);
    hipStreamSynchronize(stream1);
    hipMemPrefetchAsync(layer.rand_gpu, size * sizeof(float), GPU_DEVICE, stream2);
    hipStreamSynchronize(stream2);

    yoloswag420blazeit360noscope<<<cuda_gridsize(size), BLOCK, 0, stream1>>>(net.input_gpu, size, layer.rand_gpu, layer.probability, layer.scale);
    check_error(hipPeekAtLastError());
}

void backward_dropout_layer_gpu(dropout_layer layer, network net)
{
    if(!net.delta_gpu) return;
    int size = layer.inputs*layer.batch;

    hipStream_t stream1;
    hipStream_t stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    hipMemPrefetchAsync(net.delta_gpu, size * sizeof(float), GPU_DEVICE, stream1);
    hipStreamSynchronize(stream1);
    hipMemPrefetchAsync(layer.rand_gpu, size * sizeof(float), GPU_DEVICE, stream2);
    hipStreamSynchronize(stream2);

    yoloswag420blazeit360noscope<<<cuda_gridsize(size), BLOCK, 0, stream1>>>(net.delta_gpu, size, layer.rand_gpu, layer.probability, layer.scale);
    check_error(hipPeekAtLastError());
}
