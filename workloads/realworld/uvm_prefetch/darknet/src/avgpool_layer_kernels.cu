#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "avgpool_layer.h"
#include "cuda_dark.h"
}

__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int k = id % c;
    id /= c;
    int b = id;

    int i;
    int out_index = (k + c*b);
    output[out_index] = 0;
    for(i = 0; i < w*h; ++i){
        int in_index = i + h*w*(k + b*c);
        output[out_index] += input[in_index];
    }
    output[out_index] /= w*h;
}

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int k = id % c;
    id /= c;
    int b = id;

    int i;
    int out_index = (k + c*b);
    for(i = 0; i < w*h; ++i){
        int in_index = i + h*w*(k + b*c);
        in_delta[in_index] += out_delta[out_index] / (w*h);
    }
}

extern "C" void forward_avgpool_layer_gpu(avgpool_layer layer, network net)
{
    size_t n = layer.c*layer.batch;

    hipStream_t stream1;
    hipStream_t stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    hipMemPrefetchAsync(net.input_gpu, n * sizeof(float), GPU_DEVICE, stream1);
    hipStreamSynchronize(stream1);
    hipMemPrefetchAsync(layer.output_gpu, n * sizeof(float), GPU_DEVICE, stream2);
    hipStreamSynchronize(stream2);

    forward_avgpool_layer_kernel<<<cuda_gridsize(n), BLOCK, 0, stream2>>>(n, layer.w, layer.h, layer.c, net.input_gpu, layer.output_gpu);
    check_error(hipPeekAtLastError());
}

extern "C" void backward_avgpool_layer_gpu(avgpool_layer layer, network net)
{
    size_t n = layer.c*layer.batch;

    hipStream_t stream1;
    hipStream_t stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    hipMemPrefetchAsync(net.delta_gpu, n * sizeof(float), GPU_DEVICE, stream1);
    hipStreamSynchronize(stream1);
    hipMemPrefetchAsync(layer.delta_gpu, n * sizeof(float), GPU_DEVICE, stream2);
    hipStreamSynchronize(stream2);

    backward_avgpool_layer_kernel<<<cuda_gridsize(n), BLOCK, 0, stream1>>>(n, layer.w, layer.h, layer.c, net.delta_gpu, layer.delta_gpu);
    check_error(hipPeekAtLastError());
}

