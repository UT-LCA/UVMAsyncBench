#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <cupti.h>
#include "../../../common/cupti_add.h"
#include "../../../common/cpu_timestamps.h"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>

using namespace nvcuda::experimental;

#define PREFETCH_COUNT 2

#ifdef TIMING
#include "timing.h"

struct timeval tv;
struct timeval tv_total_start, tv_total_end;
struct timeval tv_h2d_start, tv_h2d_end;
struct timeval tv_d2h_start, tv_d2h_end;
struct timeval tv_kernel_start, tv_kernel_end;
struct timeval tv_mem_alloc_start, tv_mem_alloc_end;
struct timeval tv_close_start, tv_close_end;
float init_time = 0, mem_alloc_time = 0, h2d_time = 0, kernel_time = 0,
      d2h_time = 0, close_time = 0, total_time = 0;
#endif

#define BLOCK_SIZE 256
#define STR_SIZE 256
#define DEVICE 0
#define HALO 1 // halo width along one direction when advancing to the next iteration

// #define BENCH_PRINT

void run(int argc, char **argv);

int rows, cols;
int *data;
int **wall;
int *result;
#define M_SEED 9
int pyramid_height;
int nblocks;

void init(int argc, char **argv)
{
    if (argc == 5)
    {
        cols = atoi(argv[1]);
        rows = atoi(argv[2]);
        pyramid_height = atoi(argv[3]);
        nblocks = atoi(argv[4]);
    }
    else
    {
        printf("Usage: dynproc row_len col_len pyramid_height\n");
        exit(0);
    }
    data = new int[rows * cols];
    wall = new int *[rows];
    for (int n = 0; n < rows; n++)
        wall[n] = data + cols * n;
    result = new int[cols];

    int seed = M_SEED;
    srand(seed);

    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            wall[i][j] = rand() % 10;
        }
    }
#ifdef BENCH_PRINT
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            printf("%d ", wall[i][j]);
        }
        printf("\n");
    }
#endif
}

void fatal(char *s)
{
    fprintf(stderr, "error: %s\n", s);
}

#define IN_RANGE(x, min, max) ((x) >= (min) && (x) <= (max))
#define CLAMP_RANGE(x, min, max) x = (x < (min)) ? min : ((x > (max)) ? max : x)
#define MIN(a, b) ((a) <= (b) ? (a) : (b))

__global__ void dynproc_kernel(
    int iteration,
    int *gpuWall,
    int *gpuSrc,
    int *gpuResults,
    int cols,
    int rows,
    int startStep,
    int border,
    int small_block_cols,
    int tile_size,
    int batches)
{
    cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
    __shared__ int prev[BLOCK_SIZE];
    __shared__ int result[BLOCK_SIZE];

    int bx = blockIdx.x;
    int tx = threadIdx.x;

    for (int b = 0; b < batches; b++)
    {
        // each block finally computes result for a small block
        // after N iterations.
        // it is the non-overlapping small blocks that cover
        // all the input data

        // calculate the boundary for the block according to
        // the boundary of its small block
        int blkX = bx * tile_size + small_block_cols * b - border;
        int blkXmax = blkX + BLOCK_SIZE - 1;

        // calculate the global thread coordination
        int xidx = blkX + tx;

        // effective range within this block that falls within
        // the valid range of the input data
        // used to rule out computation outside the boundary.
        int validXmin = (blkX < 0) ? -blkX : 0;
        int validXmax = (blkXmax > cols - 1) ? BLOCK_SIZE - 1 - (blkXmax - cols + 1) : BLOCK_SIZE - 1;

        int W = tx - 1;
        int E = tx + 1;

        W = (W < validXmin) ? validXmin : W;
        E = (E > validXmax) ? validXmax : E;

        bool isValid = IN_RANGE(tx, validXmin, validXmax);

        if (IN_RANGE(xidx, 0, cols - 1))
        {
            prev[tx] = gpuSrc[xidx];
        }
        block.sync(); // [Ronny] Added sync to avoid race on prev Aug. 14 2012
        bool computed;
        for (int i = 0; i < iteration; i++)
        {
            computed = false;
            if (IN_RANGE(tx, i + 1, BLOCK_SIZE - i - 2) &&
                isValid)
            {
                computed = true;
                int left = prev[W];
                int up = prev[tx];
                int right = prev[E];
                int shortest = MIN(left, up);
                shortest = MIN(shortest, right);
                int index = cols * (startStep + i) + xidx;
                result[tx] = shortest + gpuWall[index];
            }
            block.sync();
            if (i == iteration - 1)
                break;
            if (computed) // Assign the computation range
                prev[tx] = result[tx];
            block.sync(); // [Ronny] Added sync to avoid race on prev Aug. 14 2012
        }

        // update the global memory
        // after the last iteration, only threads coordinated within the
        // small block perform the calculation and switch on ``computed''
        if (computed)
        {
            gpuResults[xidx] = result[tx];
        }
    }

    
}

/*
   compute N time steps
*/
int calc_path(int *gpuWall, int *gpuResult[2], int rows, int cols,
              int pyramid_height, int blockCols, int borderCols, int tile_size, int batches)
{
    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid(nblocks);

    int src = 1, dst = 0;
    for (int t = 0; t < rows - 1; t += pyramid_height)
    {
        int temp = src;
        src = dst;
        dst = temp;

        int iteration = MIN(pyramid_height, rows - t - 1);
        int small_block_cols = BLOCK_SIZE - iteration * HALO * 2;
        dynproc_kernel<<<dimGrid, dimBlock>>>(
            iteration, gpuWall, gpuResult[src], gpuResult[dst],
            cols, rows, t, borderCols, small_block_cols, tile_size, batches);

        // for the measurement fairness
        hipDeviceSynchronize();
    }
    return dst;
}

extern inline __attribute__((always_inline)) unsigned long rdtsc()
{
    unsigned long a, d;

    __asm__ volatile("rdtsc"
                     : "=a"(a), "=d"(d));

    return (a | (d << 32));
}

extern inline __attribute__((always_inline)) unsigned long rdtsp()
{
    struct timespec tms;
    if (clock_gettime(CLOCK_REALTIME, &tms))
    {
        return -1;
    }
    unsigned long ns = tms.tv_sec * 1000000000;
    ns += tms.tv_nsec;
    return ns;
}

#define GPU_DEVICE 6

void GPU_argv_init()
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
    printf("setting device %d with name %s\n", GPU_DEVICE, deviceProp.name);
    hipSetDevice(GPU_DEVICE);
}

int main(int argc, char *argv[])
{
    uint64_t start_tsc = rdtsc();
    uint64_t start_tsp = rdtsp();
    printf("start_tsc %lu start_tsp %lu\n", start_tsc, start_tsp);
    GPU_argv_init();

    run(argc, argv);

    return EXIT_SUCCESS;
}

void run(int argc, char **argv)
{
    init(argc, argv);

    /* --------------- pyramid parameters --------------- */
    int borderCols = (pyramid_height)*HALO;
    int smallBlockCol = BLOCK_SIZE - (pyramid_height)*HALO * 2;
    int blockCols = cols / smallBlockCol + ((cols % smallBlockCol == 0) ? 0 : 1);

    //ruihao
    int cols_per_block = cols / nblocks;
    if (cols_per_block < BLOCK_SIZE) cols_per_block = BLOCK_SIZE;
    int batches = cols_per_block / smallBlockCol + ((cols_per_block % smallBlockCol == 0) ? 0 : 1);

    // printf("pyramidHeight: %d\ngridSize: [%d]\nborder:[%d]\nblockSize: %d\nblockGrid:[%d]\ntargetBlock:[%d]\n",
    //        pyramid_height, cols, borderCols, BLOCK_SIZE, blockCols, smallBlockCol);
    printf("pyramidHeight: %d\ngridSize: [%d]\nborder:[%d]\nblockSize: %d\nblockGrid:[%d]\ntargetBlock:[%d]\n",
           pyramid_height, cols, borderCols, BLOCK_SIZE, nblocks, smallBlockCol);

    int *gpuWall, *gpuResult[2];
    int size = rows * cols;

    initTrace();
    startCPU();

    hipMalloc((void **)&gpuResult[0], sizeof(int) * cols);
    hipMalloc((void **)&gpuResult[1], sizeof(int) * cols);
    hipMemcpy(gpuResult[0], data, sizeof(int) * cols, hipMemcpyHostToDevice);
    hipMalloc((void **)&gpuWall, sizeof(int) * (size - cols));
    hipMemcpy(gpuWall, data + cols, sizeof(int) * (size - cols), hipMemcpyHostToDevice);

#ifdef TIMING
    gettimeofday(&tv_kernel_start, NULL);
#endif

    // int final_ret = calc_path(gpuWall, gpuResult, rows, cols,
    //                           pyramid_height, blockCols, borderCols);
    int final_ret = calc_path(gpuWall, gpuResult, rows, cols,
                              pyramid_height, blockCols, borderCols, cols_per_block, batches);

#ifdef TIMING
    gettimeofday(&tv_kernel_end, NULL);
    tvsub(&tv_kernel_end, &tv_kernel_start, &tv);
    kernel_time += tv.tv_sec * 1000.0 + (float)tv.tv_usec / 1000.0;
#endif

    hipMemcpy(result, gpuResult[final_ret], sizeof(int) * cols, hipMemcpyDeviceToHost);

#ifdef BENCH_PRINT
    for (int i = 0; i < cols; i++)
        printf("%d ", data[i]);
    printf("\n");
    for (int i = 0; i < cols; i++)
        printf("%d ", result[i]);
    printf("\n");
#endif

    hipFree(gpuWall);
    hipFree(gpuResult[0]);
    hipFree(gpuResult[1]);

    endCPU();
    finiTrace();

    delete[] data;
    delete[] wall;
    delete[] result;

#ifdef TIMING
    printf("Exec: %f\n", kernel_time);
#endif
}
