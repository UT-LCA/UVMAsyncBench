#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
//#include <cutil.h>
// includes CUDA
#include <hip/hip_runtime.h>
// includes, kernels
#include "ordergraph_kernel.cu"
;

#include "../../../common/cpu_timestamps.h"
#include "../../../common/cupti_add.h"

const int HIGHEST = 3;
int taskperthr = 1;
int sizepernode;
int ITER = 100;

// global var
float preScore = -99999999999.0f;
float score = 0.0;
float maxScore[HIGHEST] = {-999999999.0f};
bool orders[NODE_N][NODE_N];
bool preOrders[NODE_N][NODE_N];
bool preGraph[NODE_N][NODE_N];
bool bestGraph[HIGHEST][NODE_N][NODE_N];
bool graph[NODE_N][NODE_N];
// float prior[NODE_N][NODE_N];
float *localscore, *D_localscore, *D_Score, *scores;
float *LG;
bool *D_parent;
int *D_resP, *parents;

void initial();  // initial orders and data
int genOrders(); // swap
int ConCore();   // discard new order or not
bool getparent(int *bit, int *pre, int posN, int *parent, int *parN,
               int time);    // get every possible set of parents for a node
void incr(int *bit, int n);  // binary code increases 1 each time
void incrS(int *bit, int n); // STATE_N code increases 1 each time
bool getState(
    int parN, int *state,
    int time); // get every possible combination of state for a parent set
float logGamma(int N); // log and gamma
float findBestGraph();
void genScore();
int convert(int *parent, int parN);
void sortGraph();
void swap(int a, int b);
void Pre_logGamma();
int findindex(int *arr, int size);
int C(int n, int a);

FILE *fpout;

extern inline __attribute__((always_inline)) unsigned long rdtsc() {
  unsigned long a, d;

  __asm__ volatile("rdtsc" : "=a"(a), "=d"(d));

  return (a | (d << 32));
}

extern inline __attribute__((always_inline)) unsigned long rdtsp() {
  struct timespec tms;
  if (clock_gettime(CLOCK_REALTIME, &tms)) {
    return -1;
  }
  unsigned long ns = tms.tv_sec * 1000000000;
  ns += tms.tv_nsec;
  return ns;
}

#define GPU_DEVICE 6

void GPU_argv_init() {
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
  printf("setting device %d with name %s\n", GPU_DEVICE, deviceProp.name);
  hipSetDevice(GPU_DEVICE);
}

int main(int argc, char *argv[]) {
  /*
      for(i=0;i<NODE_N;i++){
              for(j=0;j<NODE_N;j++)
              prior[i][j]=0.5;
      }
  */
  uint64_t start_tsc = rdtsc();
  uint64_t start_tsp = rdtsp();
  printf("start_tsc %lu start_tsp %lu\n", start_tsc, start_tsp);
  int i, j, c = 0, tmp, a, b;
  float tmpd;
  #ifdef DATA_25
  char name[20] = "25.out";
  #endif
  #ifdef DATA_30
  char name[20] = "30.out";
  #endif
  #ifdef DATA_40
  char name[20] = "40.out";
  #endif
  #ifdef DATA_45
  char name[20] = "45.out";
  #endif
  #ifdef DATA_50
  char name[20] = "50.out";
  #endif
  #ifdef DATA_125
  char name[20] = "125.out";
  #endif

  fpout = fopen(name, "w");

  clock_t start, finish, total = 0, pre1, pre2;
  hipDeviceSynchronize();

  printf("NODE_N=%d\nInitialization...\n", NODE_N);
  pre1 = clock();

  srand(time(NULL));
  initial();

  GPU_argv_init();
  initTrace();
  startCPU();

  genScore();
  pre2 = clock();
  printf("OK, begin to generate orders.\n");

  i = 0;
  while (i != ITER) {

    start = clock();

    i++;
    score = 0;

    for (a = 0; a < NODE_N; a++) {
      for (j = 0; j < NODE_N; j++) {
        orders[a][j] = preOrders[a][j];
      }
    }

    tmp = rand() % 6;
    for (j = 0; j < tmp; j++)
      genOrders();

    score = findBestGraph();

    finish = clock();
    total += finish - start;

    ConCore();

    // store the top HIGHEST highest orders
    if (c < HIGHEST) {
      tmp = 1;
      for (j = 0; j < c; j++) {
        if (maxScore[j] == preScore) {
          tmp = 0;
        }
      }
      if (tmp != 0) {
        maxScore[c] = preScore;
        for (a = 0; a < NODE_N; a++) {
          for (b = 0; b < NODE_N; b++) {
            bestGraph[c][a][b] = preGraph[a][b];
          }
        }
        c++;
      }

    } else if (c == HIGHEST) {
      sortGraph();
      c++;
    } else {

      tmp = 1;
      for (j = 0; j < HIGHEST; j++) {
        if (maxScore[j] == preScore) {
          tmp = 0;
          break;
        }
      }
      if (tmp != 0 && preScore > maxScore[HIGHEST - 1]) {
        maxScore[HIGHEST - 1] = preScore;
        for (a = 0; a < NODE_N; a++) {
          for (b = 0; b < NODE_N; b++) {
            bestGraph[HIGHEST - 1][a][b] = preGraph[a][b];
          }
        }
        b = HIGHEST - 1;
        for (a = HIGHEST - 2; a >= 0; a--) {
          if (maxScore[b] > maxScore[a]) {
            swap(a, b);
            tmpd = maxScore[a];
            maxScore[a] = maxScore[b];
            maxScore[b] = tmpd;
            b = a;
          }
        }
      }
    }

  } // endwhile

  hipHostFree(localscore);
  hipFree(D_localscore);
  hipFree(D_parent);

  hipHostFree(scores);
  hipHostFree(parents);
  hipFree(D_Score);
  hipFree(D_resP);

  /*
          for(j=0;j<HIGHEST;j++){
                          fprintf(fpout,"score:%f\n",maxScore[j]);
                          fprintf(fpout,"Best Graph:\n");
                          for(int a=0;a<NODE_N;a++){
                                  for(int b=0;b<NODE_N;b++){
                                          fprintf(fpout,"%d
     ",bestGraph[j][a][b]);
                                  }
                                  fprintf(fpout,"%\n");
                          }
                          fprintf(fpout,"--------------------------------------------------------------------\n");
                  }*/
  endCPU();
  finiTrace();
  fprintf(fpout, "Duration per interation is %f seconds.\n",
          ((float)total / ITER) / CLOCKS_PER_SEC);
  fprintf(fpout, "Total duration is %f seconds.\n",
          (float)(pre2 - pre1 + total) / CLOCKS_PER_SEC);
  fprintf(fpout, "Preprocessing duration is %f seconds.\n",
          (float)(pre2 - pre1) / CLOCKS_PER_SEC);

  printf("Duration per interation is %f seconds.\n",
         ((float)total / ITER) / CLOCKS_PER_SEC);
  printf("Total duration is %f seconds.\n",
         (float)(pre2 - pre1 + total) / CLOCKS_PER_SEC);
  printf("Preprocessing duration is %f seconds.\n",
         (float)(pre2 - pre1) / CLOCKS_PER_SEC);

  return 0;
}

void sortGraph() {
  float max = -99999999999999;
  int maxi, i, j;
  float tmp;

  for (j = 0; j < HIGHEST - 1; j++) {
    max = maxScore[j];
    maxi = j;
    for (i = j + 1; i < HIGHEST; i++) {
      if (maxScore[i] > max) {
        max = maxScore[i];
        maxi = i;
      }
    }

    swap(j, maxi);
    tmp = maxScore[j];
    maxScore[j] = max;
    maxScore[maxi] = tmp;
  }
}

void swap(int a, int b) {
  int i, j;
  bool tmp;

  for (i = 0; i < NODE_N; i++) {
    for (j = 0; j < NODE_N; j++) {

      tmp = bestGraph[a][i][j];
      bestGraph[a][i][j] = bestGraph[b][i][j];
      bestGraph[b][i][j] = tmp;
    }
  }
}

void initial() {
  int i, j, tmp, a, b, r;
  bool tmpd;
  tmp = 1;
  for (i = 1; i <= 4; i++) {
    tmp += C(NODE_N - 1, i);
  }
  sizepernode = tmp;
  tmp *= NODE_N;

  hipHostMalloc((void **)&localscore, tmp * sizeof(float));

  for (i = 0; i < tmp; i++)
    localscore[i] = 0;

  for (i = 0; i < NODE_N; i++) {
    for (j = 0; j < NODE_N; j++)
      orders[i][j] = 0;
  }
  for (i = 0; i < NODE_N; i++) {
    for (j = 0; j < i; j++)
      orders[i][j] = 1;
  }
  r = rand() % 10000;
  for (i = 0; i < r; i++) {
    a = rand() % NODE_N;
    b = rand() % NODE_N;
    for (j = 0; j < NODE_N; j++) {
      tmpd = orders[j][a];
      orders[j][a] = orders[j][b];
      orders[j][b] = tmpd;
    }

    for (j = 0; j < NODE_N; j++) {
      tmpd = orders[a][j];
      orders[a][j] = orders[b][j];
      orders[b][j] = tmpd;
    }
  }

  for (i = 0; i < NODE_N; i++) {
    for (j = 0; j < NODE_N; j++) {
      preOrders[i][j] = orders[i][j];
    }
  }
}

// generate ramdom order
int genOrders() {

  int a, b, j;
  bool tmp;
  a = rand() % NODE_N;
  b = rand() % NODE_N;

  for (j = 0; j < NODE_N; j++) {
    tmp = orders[a][j];
    orders[a][j] = orders[b][j];
    orders[b][j] = tmp;
  }
  for (j = 0; j < NODE_N; j++) {
    tmp = orders[j][a];
    orders[j][a] = orders[j][b];
    orders[j][b] = tmp;
  }

  return 1;
}

// decide leave or discard an order
int ConCore() {
  int i, j;
  float tmp;
  tmp = log((rand() % 100000) / 100000.0);
  if (tmp < (score - preScore)) {

    for (i = 0; i < NODE_N; i++) {
      for (j = 0; j < NODE_N; j++) {
        preOrders[i][j] = orders[i][j];
        preGraph[i][j] = graph[i][j];
      }
    }
    preScore = score;

    return 1;
  }

  return 0;
}

void genScore() {
  int *D_data;
  float *D_LG;
  dim3 grid(sizepernode / 256 + 1, 1, 1);
  dim3 threads(256, 1, 1);

  Pre_logGamma();
  // cudaPrintfInit();
  hipMalloc((void **)&D_data, NODE_N * DATA_N * sizeof(int));
  hipMalloc((void **)&D_localscore, NODE_N * sizepernode * sizeof(float));
  hipMalloc((void **)&D_LG, (DATA_N + 2) * sizeof(float));
  hipMemset(D_localscore, 0.0, NODE_N * sizepernode * sizeof(float));
  hipMemcpy(D_data, data, NODE_N * DATA_N * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(D_LG, LG, (DATA_N + 2) * sizeof(float), hipMemcpyHostToDevice);
  genScoreKernel<<<grid, threads>>>(sizepernode, D_localscore, D_data, D_LG);
  hipDeviceSynchronize();
  hipMemcpy(localscore, D_localscore, NODE_N * sizepernode * sizeof(float),
             hipMemcpyDeviceToHost);

  // cudaPrintfDisplay(stdout, true);
  // cudaPrintfEnd();

  hipHostFree(LG);
  hipFree(D_LG);
  hipFree(D_data);

  hipHostMalloc((void **)&scores,
                 (sizepernode / (256 * taskperthr) + 1) * sizeof(float));
  hipHostMalloc((void **)&parents,
                 (sizepernode / (256 * taskperthr) + 1) * 4 * sizeof(int));
  hipMalloc((void **)&D_Score,
             (sizepernode / (256 * taskperthr) + 1) * sizeof(float));
  hipMalloc((void **)&D_parent, NODE_N * sizeof(bool));
  hipMalloc((void **)&D_resP,
             (sizepernode / (256 * taskperthr) + 1) * 4 * sizeof(int));
}

int convert(int *parent, int parN) {
  int i, j, w = 1, tmp = 0;
  j = 0;
  for (i = 0; parN > 0 && i <= parent[parN - 1]; i++) {
    if (parent[j] == i) {
      j++;
      tmp += w;
    }
    w *= 2;
  }

  return tmp;
}

void Pre_logGamma() {

  hipHostMalloc((void **)&LG, (DATA_N + 2) * sizeof(float));

  LG[1] = log(1.0);
  float i;
  for (i = 2; i <= DATA_N + 1; i++) {
    LG[(int)i] = LG[(int)i - 1] + log((float)i);
  }
}

void incr(int *bit, int n) {

  bit[n]++;
  if (bit[n] >= 2) {
    bit[n] = 0;
    incr(bit, n + 1);
  }

  return;
}

void incrS(int *bit, int n) {

  bit[n]++;
  if (bit[n] >= STATE_N) {
    bit[n] = 0;
    incr(bit, n + 1);
  }

  return;
}

bool getState(int parN, int *state, int time) {
  int j = 1;

  j = pow(STATE_N, (float)parN) - 1;

  if (time > j)
    return false;

  if (time >= 1)
    incrS(state, 0);

  return true;
}

bool getparent(int *bit, int *pre, int posN, int *parent, int *parN, int time) {
  int i, j = 1;

  *parN = 0;
  if (time == 0)
    return true;

  for (i = 0; i < posN; i++) {
    j = j * 2;
  }
  j--;

  if (time > j)
    return false;

  incr(bit, 0);

  for (i = 0; i < posN; i++) {
    if (bit[i] == 1) {
      parent[(*parN)++] = pre[i];
    }
  }

  return true;
}

float findBestGraph() {
  float bestls = -99999999;
  int bestparent[5];
  int bestpN, total;
  int node, index;
  int pre[NODE_N] = {0};
  int parent[NODE_N] = {0};
  int posN = 0, i, j, parN, tmp, k, l;
  float ls = -99999999999, score = 0;
  int blocknum;

  for (i = 0; i < NODE_N; i++)
    for (j = 0; j < NODE_N; j++)
      graph[i][j] = 0;

  for (node = 0; node < NODE_N; node++) {

    bestls = -99999999;
    posN = 0;

    for (i = 0; i < NODE_N; i++) {
      if (orders[node][i] == 1) {
        pre[posN++] = i;
      }
    }

    if (posN >= 0) {
      total = C(posN, 4) + C(posN, 3) + C(posN, 2) + posN + 1;
      taskperthr = 1;
      blocknum = total / (256 * taskperthr) + 1;

      int nbatches = MIN_NBATCHES;

      int blocknum_max = total / (BLOCK_SIZE * MIN_NBATCHES * taskperthr) + 1;
      if (blocknum_max >= MAX_NBLOCKS) {
        blocknum = MAX_NBLOCKS;
        nbatches = (total + 1) / (BLOCK_SIZE * MAX_NBLOCKS * taskperthr);
      } else {
        blocknum = blocknum_max;
      }

      hipMemset(D_resP, 0, blocknum * 4 * sizeof(int));
      hipMemset(D_Score, -999999.0, blocknum * nbatches * sizeof(float));
      hipMemcpy(D_parent, orders[node], NODE_N * sizeof(bool),
                 hipMemcpyHostToDevice);

      computeKernel<<<blocknum, 256>>>(
          taskperthr, sizepernode, D_localscore, D_parent, node, total, D_Score,
          D_resP, nbatches);
      hipDeviceSynchronize();
      hipMemcpy(parents, D_resP, blocknum * 4 * sizeof(int),
                 hipMemcpyDeviceToHost);
      hipMemcpy(scores, D_Score, blocknum * sizeof(float),
                 hipMemcpyDeviceToHost);

      for (i = 0; i < blocknum * nbatches; i++) {

        if (scores[i] > bestls) {

          bestls = scores[i];

          parN = 0;
          for (tmp = 0; tmp < 4; tmp++) {
            if (parents[i * 4 + tmp] < 0)
              break;

            bestparent[tmp] = parents[i * 4 + tmp];

            parN++;
          }

          bestpN = parN;
        }
      }
    } else {
      if (posN >= 4) {
        for (i = 0; i < posN; i++) {
          for (j = i + 1; j < posN; j++) {
            for (k = j + 1; k < posN; k++) {
              for (l = k + 1; l < posN; l++) {
                parN = 4;
                if (pre[i] > node)
                  parent[1] = pre[i];
                else
                  parent[1] = pre[i] + 1;
                if (pre[j] > node)
                  parent[2] = pre[j];
                else
                  parent[2] = pre[j] + 1;
                if (pre[k] > node)
                  parent[3] = pre[k];
                else
                  parent[3] = pre[k] + 1;
                if (pre[l] > node)
                  parent[4] = pre[l];
                else
                  parent[4] = pre[l] + 1;

                index = findindex(parent, parN);
                index += sizepernode * node;
                ls = localscore[index];

                if (ls > bestls) {
                  bestls = ls;
                  bestpN = parN;
                  for (tmp = 0; tmp < parN; tmp++)
                    bestparent[tmp] = parent[tmp + 1];
                }
              }
            }
          }
        }
      }

      if (posN >= 3) {
        for (i = 0; i < posN; i++) {
          for (j = i + 1; j < posN; j++) {
            for (k = j + 1; k < posN; k++) {

              parN = 3;
              if (pre[i] > node)
                parent[1] = pre[i];
              else
                parent[1] = pre[i] + 1;
              if (pre[j] > node)
                parent[2] = pre[j];
              else
                parent[2] = pre[j] + 1;
              if (pre[k] > node)
                parent[3] = pre[k];
              else
                parent[3] = pre[k] + 1;

              index = findindex(parent, parN);
              index += sizepernode * node;
              ls = localscore[index];

              if (ls > bestls) {
                bestls = ls;
                bestpN = parN;
                for (tmp = 0; tmp < parN; tmp++)
                  bestparent[tmp] = parent[tmp + 1];
              }
            }
          }
        }
      }

      if (posN >= 2) {
        for (i = 0; i < posN; i++) {
          for (j = i + 1; j < posN; j++) {

            parN = 2;
            if (pre[i] > node)
              parent[1] = pre[i];
            else
              parent[1] = pre[i] + 1;
            if (pre[j] > node)
              parent[2] = pre[j];
            else
              parent[2] = pre[j] + 1;

            index = findindex(parent, parN);
            index += sizepernode * node;
            ls = localscore[index];

            if (ls > bestls) {
              bestls = ls;
              bestpN = parN;
              for (tmp = 0; tmp < parN; tmp++)
                bestparent[tmp] = parent[tmp + 1];
            }
          }
        }
      }

      if (posN >= 1) {
        for (i = 0; i < posN; i++) {

          parN = 1;
          if (pre[i] > node)
            parent[1] = pre[i];
          else
            parent[1] = pre[i] + 1;

          index = findindex(parent, parN);
          index += sizepernode * node;
          ls = localscore[index];

          if (ls > bestls) {
            bestls = ls;
            bestpN = parN;
            for (tmp = 0; tmp < parN; tmp++)
              bestparent[tmp] = parent[tmp + 1];
          }
        }
      }

      parN = 0;
      index = sizepernode * node;

      ls = localscore[index];

      if (ls > bestls) {
        bestls = ls;
        bestpN = 0;
      }
    }
    if (bestls > -99999999) {

      for (i = 0; i < bestpN; i++) {
        if (bestparent[i] < node)
          graph[node][bestparent[i] - 1] = 1;
        else
          graph[node][bestparent[i]] = 1;
      }
      score += bestls;
    }
  }

  return score;
}

int findindex(int *arr, int size) { // reminder: arr[0] has to be 0 && size ==
                                    // array size-1 && index start from 0
  int i, j, index = 0;

  for (i = 1; i < size; i++) {
    index += C(NODE_N - 1, i);
  }

  for (i = 1; i <= size - 1; i++) {
    for (j = arr[i - 1] + 1; j <= arr[i] - 1; j++) {
      index += C(NODE_N - 1 - j, size - i);
    }
  }

  index += arr[size] - arr[size - 1];

  return index;
}

int C(int n, int a) {
  int i, res = 1, atmp = a;

  for (i = 0; i < atmp; i++) {
    res *= n;
    n--;
  }

  for (i = 0; i < atmp; i++) {
    res /= a;
    a--;
  }

  return res;
}